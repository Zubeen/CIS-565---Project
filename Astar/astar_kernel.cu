#include "hip/hip_runtime.h"

#ifndef _ASTAR_KERNEL_H_
#define _ASTAR_KERNEL_H_

#include "cutil_math.h"
#include <stdio.h>

#define BLOCK_SIZE 64

#define FOUND 1
#define NONEXISTANT 0


//Astarsearch<<<num_blocks,BLOCK_SIZE>>>(&map[0][0],mapX, mapY, startX, startY, goalX, goalY, path, isComplete); 

__global__ void Astarsearch(int* map, int mapX, int mapY, int *startX, int *startY, int *goalX, int *goalY, int *pathX, int *pathY, int *isComplete)
{
	int thid = blockIdx.x*blockDim.x + threadIdx.x;  

	int openList[mayX*mapY+2]; // 1D array holding ID# of open list items
	int whichList[mapX+1][mapY+1]; // record whether a cell is on open (0) or closed(1) list
	int openX[mapX*mapY+2], openY[mapX*mapY+2]; //1D arrays to store X and Y location of each node on the open list
	int parentX[mapX+1][mapY+1],parentY[mapX+1][mapY+1]; //2D arrays to store parents of each cell
	
	int Fcost[mapX*mapY+2]; // 1D array to store cost of cell on open list
	int Hcost[mapX*mapY+2]; // 1D array to store Heuristic cost of cell on open list
	int Gcost[mapX+1][mapY+1]; // 1D array to store cost of cell on open list

	int pathLength=0;
	

	int 0=0, parentXval=0, parentYval=0,
	a=0, b=0, m=0, u=0, v=0, temp=0, corner=0, numberOfOpenListItems=0,
	addedGCost=0, tempGcost = 0, path = 0,
	tempx, newOpenListItemID=0;

	if (startX == targetX && startY == targetY)
	{
		isComplete=0;
	}
	else if (map[targetX][targetY] == 1) //If target square is an obstacle, return that it's a nonexistent path.
	{
		isComplete=-1;
	}
	else
	{
		Gcost[startX][startY] = 0; //set starting square's G value to 0
		
		//Add the starting location to the open list of squares to be checked.
		numberOfOpenListItems = 1;
		openList[1] = 1;//assign it as the top (and currently only) item in the open list, which is maintained as a binary heap (explained below)
		openX[1] = startX[thid] ; openY[1] = startY[thid];

		//Until a path is found or deemed nonexistent.
		do
		{
		
			//If the open list is not empty, take the first cell off of the list. (This is the lowest F cost cell on the open list)
			
			if (numberOfOpenListItems != 0)
			{
				//Pop the first item off the open list.
				parentXval = openX[openList[1]];
				parentYval = openY[openList[1]]; //record cell coordinates of the item
				whichList[parentXval][parentYval] = 1;//add the item to the closed list

				//	Open List = Binary Heap: Delete this item from the open list, which is maintained as a binary heap. For more information on binary heaps, see:
				numberOfOpenListItems = numberOfOpenListItems - 1;//reduce number of open list items by 1	
		
				//	Delete the top item in binary heap and reorder the heap, with the lowest F cost item rising to the top.
				openList[1] = openList[numberOfOpenListItems+1];//move the last item in the heap up to slot #1
				v = 1;

				//	Re-make Heap
				do
				{
					u = v;		
					if (2*u+1 <= numberOfOpenListItems) //if both children exist
					{
	 					//Check if the F cost of the parent is greater than each child.
						//Select the lowest of the two children.
						if (Fcost[openList[u]] >= Fcost[openList[2*u]]) 
							v = 2*u;
						if (Fcost[openList[v]] >= Fcost[openList[2*u+1]]) 
							v = 2*u+1;		
					}
					else
					{
						if (2*u <= numberOfOpenListItems) //if only child #1 exists
						{
	 						//Check if the F cost of the parent is greater than child #1	
							if (Fcost[openList[u]] >= Fcost[openList[2*u]]) 
								v = 2*u;
						}
					}

					if (u != v) //if parent's F is > one of its children, swap them
					{
						temp = openList[u];
						openList[u] = openList[v];
						openList[v] = temp;			
					}
					else
						break; //otherwise, exit loop
				}
				while (1); // Remake Heap ends


				// Check the adjacent squares. Add these adjacent child squares to the open list for later consideration if appropriate
				
				for (b = parentYval-1; b <= parentYval+1; b++)
				{
					for (a = parentXval-1; a <= parentXval+1; a++)
					{

						//	If not off the map (do this first to avoid array out-of-bounds errors)
						if (a != -1 && b != -1 && a != mapWidth && b != mapHeight)
						{

							//	If not already on the closed list 			
							if (whichList[a][b] != 1)
							{ 
								//	If not a wall/obstacle square.
								if (map [a][b] != 1) 
								{ 
									//	Don't cut across corners
									corner = 0;	
									if (a == parentXval-1) 
									{
										if (b == parentYval-1)
										{
											if (map[parentXval-1][parentYval] == 1 || map[parentXval][parentYval-1] == 1) 
												corner = 1;
										}
										else if (b == parentYval+1)
										{	
											if (map[parentXval][parentYval+1] == 1 || map[parentXval-1][parentYval] == 1) 
											corner = 1; 
										}
									}
									else if (a == parentXval+1)
									{
										if (b == parentYval-1)
										{
											if (map[parentXval][parentYval-1] == 1 || map[parentXval+1][parentYval] == 1) 
												corner = 1;
										}
										else if (b == parentYval+1)
										{
											if (map[parentXval+1][parentYval] == 1 || map[parentXval][parentYval+1] == 1)
												corner = 1;
										}
									}	
									if (corner == 0)
									{
										//	If not already on the open list, add it to the open list.			
										if (whichList[a][b] != 0) 
										{	
											//Create a new open list item in the binary heap.
											newOpenListItemID = newOpenListItemID + 1; //each new item has a unique ID #
											m = numberOfOpenListItems+1;
											openList[m] = newOpenListItemID;//place the new open list item (actually, its ID#) at the bottom of the heap
											openX[newOpenListItemID] = a;
											openY[newOpenListItemID] = b;//record the x and y coordinates of the new item

											//Figure out its G cost
											if (abs(a-parentXval) == 1 && abs(b-parentYval) == 1)
												addedGCost = 15;//cost of going to diagonal squares	
											else	
												addedGCost = 10;//cost of going to non-diagonal squares				
										
											Gcost[a][b] = Gcost[parentXval][parentYval] + addedGCost;

											//Figure out its H and F costs and parent
											Hcost[openList[m]] = 10*(abs(a - targetX) + abs(b - targetY));
											Fcost[openList[m]] = Gcost[a][b] + Hcost[openList[m]];
											parentX[a][b] = parentXval ; parentY[a][b] = parentYval;	

										//Move the new open list item to the proper place in the binary heap.
											while (m != 1) //While item hasn't bubbled to the top (m=1)	
											{
												//Check if child's F cost is < parent's F cost. If so, swap them.	
												if (Fcost[openList[m]] <= Fcost[openList[m/2]])
												{
													temp = openList[m/2];
													openList[m/2] = openList[m];
													openList[m] = temp;
													m = m/2;
												}
												else
													break;
											}
											numberOfOpenListItems = numberOfOpenListItems+1;//add one to the number of items in the heap

											//Change whichList to show that the new item is on the open list.
											whichList[a][b] = 0;
										}

										//  If adjacent cell is already on the open list, check to see if this 
										//	path to that cell from the starting location is a better one. 
										//	If so, change the parent of the cell and its G and F costs.	
										else //If whichList(a,b) = 0
										{
											//Figure out the G cost of this possible new path
											if (abs(a-parentXval) == 1 && abs(b-parentYval) == 1)
												addedGCost = 15;//cost of going to diagonal tiles	
											else	
												addedGCost = 10;//cost of going to non-diagonal tiles				
									
											tempGcost = Gcost[parentXval][parentYval] + addedGCost;
	
											//If this path is shorter (G cost is lower) then change
											//the parent cell, G cost and F cost. 		
											if (tempGcost < Gcost[a][b]) //if G cost is less,
											{
												parentX[a][b] = parentXval; //change the square's parent
												parentY[a][b] = parentYval;
												Gcost[a][b] = tempGcost;//change the G cost			
											
												//Because changing the G cost also changes the F cost, if
												//the item is on the open list we need to change the item's
												//recorded F cost and its position on the open list to make
												//sure that we maintain a properly ordered open list.
												for (int x = 1; x <= numberOfOpenListItems; x++) //look for the item in the heap
												{
													if (openX[openList[x]] == a && openY[openList[x]] == b) //item found
													{
														Fcost[openList[x]] = Gcost[a][b] + Hcost[openList[x]];//change the F cost
														//See if changing the F score bubbles the item up from it's current location in the heap
														m = x;
														while (m != 1) //While item hasn't bubbled to the top (m=1)	
														{
															//Check if child is < parent. If so, swap them.	
															if (Fcost[openList[m]] < Fcost[openList[m/2]])
															{
																temp = openList[m/2];
																openList[m/2] = openList[m];
																openList[m] = temp;
																m = m/2;
															}
															else
																break;
														} 
														break; //exit for x = loop
													} //If openX(openList(x)) = a
												} //For x = 1 To numberOfOpenListItems
											}//If tempGcost < Gcost(a,b)
										}//else If whichList(a,b) = 0	
									}//If not cutting a corner
								}//If not a wall/obstacle square.
							}//If not already on the closed list 
						}//If not off the map
					}//for (a = parentXval-1; a <= parentXval+1; a++)
				}//for (b = parentYval-1; b <= parentYval+1; b++)
			}//if (numberOfOpenListItems != 0)
			else
			{
				isComplete[thid]=-1;
				break;	
			}  
		
		
			//If target is added to open list then path has been found.
			if (whichList[targetX][targetY] == 0)
			{
				path = FOUND;
				break;
			}
		}
		while (1);//Do until path is found or deemed nonexistent

		//Save the path if it exists.
		if (path == FOUND)
		{
			// Working backwards from the target to the starting location by checking
			//	each cell's parent, figure out the length of the path.
			int k=0
			pathX[thid][0] = targetX; pathY[thid][0] = targetY;
			do
			{
				k++;
				//Look up the parent of the current cell.	
				pathX[thid][k]  = parentX[pathX[thid][k-1]][pathY[thid][k-1]];		
				pathY[thid][k]  = parentY[pathX[thid][k-1]][pathY[thid][k-1]];		
		
				//Figure out the path length
				pathLength = pathLength + 1;
			}
			while (pathX[thid][k] != startX[thid] || pathY[thid][k] != startY[thid]);
		
			isComplete[thid]=pathLength;
		}
	}

}

extern "C"
void astar(int* map, int mapX, int mapY, int N)
{
	//The start and the goal for the parallel A* searches will be assigned by the "CPU" code
	int *startX;int *startY;
	hipMalloc((void**)&startX, N*sizeof(int));hipMalloc((void**)&startY, N*sizeof(int));

	int *goalX;int *goalY;
	hipMalloc((void**)&goalX, N*sizeof(int));hipMalloc((void**)&goalY, N*sizeof(int));

	int *pathX;
	hipMalloc((void**)&path, N*sizeof(int));hipMalloc((void**)&path, N*sizeof(int));
	int *pathY;
	hipMalloc((void**)&pathY, N*sizeof(int));hipMalloc((void**)&pathY, N*sizeof(int));

	int *isComplete;
	hipMalloc((void**)&isComplete, N*sizeof(int));hipMalloc((void**)&isComplete, N*sizeof(int));

	int num_blocks =0;
	num_blocks = N/BLOCK_SIZE;
	Astarsearch<<<num_blocks,BLOCK_SIZE>>>(&map[0][0],mapX, mapY, startX, startY, goalX, goalY, pathX, pathY isComplete); //do map, mapX, mapY have to be on device ???
	
}

#endif // #ifndef _ASTAR_KERNEL_H_
